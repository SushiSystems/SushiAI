#include "hip/hip_runtime.h"
#include "ops.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <memory>
#include <algorithm>
#include <cassert>

namespace SushiAI
{
    // ------ CUDA KERNEL TANIMLARI ------

    __global__ void addKernel(const float* a, const float* b, float* result, int size)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < size)
            result[i] = a[i] + b[i];
    }

    __global__ void reluKernel(const float* input, float* output, int size)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < size)
            output[i] = fmaxf(0.0f, input[i]);
    }

    __global__ void sigmoidKernel(const float* input, float* output, int size)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < size)
            output[i] = 1.0f / (1.0f + expf(-input[i]));
    }

    __global__ void tanhKernel(const float* input, float* output, int size)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < size)
            output[i] = tanhf(input[i]);
    }

    __global__ void softmaxExpShiftKernel(const float* input, float* output, float shift, int size)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < size)
            output[i] = expf(input[i] - shift);
    }

    __global__ void softmaxNormalizeKernel(float* data, float sumExp, int size)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;

        if (i < size)
            data[i] /= sumExp;
    }
    __global__ void matmulKernel(const float* A, const float* B, float* C, int M, int K, int N)
    {
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;

        if (row < M && col < N)
        {
            float value = 0;

            for (int i = 0; i < K; ++i)
                value += A[row * K + i] * B[i * N + col];

            C[row * N + col] = value;
        }
    }

    // ------- HELPER -------
    inline dim3 getGrid(int size, int threads = 256)
    {
        return dim3((size + threads - 1) / threads);
    }

    // -------- AUTOGRAD+POINTER CUDA OPS --------

    std::shared_ptr<Tensor> add_cuda(const std::shared_ptr<Tensor>& a, const std::shared_ptr<Tensor>& b)
    {
        assert(a -> getShape() == b -> getShape());

        int size = static_cast<int>(a -> getData().size());

        bool req_grad = a -> requires_grad || b -> requires_grad;
        auto result = std::make_shared<Tensor>(a -> getShape(), 0.0f, req_grad);

        a -> copyToDevice();
        b -> copyToDevice();
        result -> allocateDevice();

        addKernel << <getGrid(size), 256 >> > (a -> devicePtr(), b -> devicePtr(), result -> devicePtr(), size);
        hipDeviceSynchronize();

        result -> copyToHost();

        if (req_grad)
        {
            auto a_ptr = a -> shared_from_this();
            auto b_ptr = b -> shared_from_this();

            auto result_ptr = result;
            result -> setGradFn([a_ptr, b_ptr, result_ptr]() 
            {
                for (size_t i = 0; i < result_ptr -> grad.size(); ++i)
                {
                    if (a_ptr -> requires_grad)
                        a_ptr -> grad[i] += result_ptr -> grad[i];
                    if (b_ptr -> requires_grad)
                        b_ptr -> grad[i] += result_ptr -> grad[i];
                }
            }, { a_ptr, b_ptr });
        }

        return result;
    }

    std::shared_ptr<Tensor> relu_cuda(const std::shared_ptr<Tensor>& t)
    {
        int size = static_cast<int>(t -> getData().size());
        auto result = std::make_shared<Tensor>(t -> getShape(), 0.0f, t -> requires_grad);

        t -> copyToDevice();
        result -> allocateDevice();

        reluKernel << <getGrid(size), 256 >> > (t -> devicePtr(), result -> devicePtr(), size);
        hipDeviceSynchronize();

        result -> copyToHost();

        if (t -> requires_grad)
        {
            auto t_ptr = t -> shared_from_this();

            auto result_ptr = result;
            result -> setGradFn([t_ptr, result_ptr]() 
            {
                for (size_t i = 0; i < result_ptr -> grad.size(); ++i)
                    t_ptr->grad[i] += (result_ptr -> data[i] > 0 ? 1.0f : 0.0f) * result_ptr -> grad[i];
            }, { t_ptr });
        }

        return result;
    }

    std::shared_ptr<Tensor> sigmoid_cuda(const std::shared_ptr<Tensor>& t)
    {
        int size = static_cast<int>(t -> getData().size());
        auto result = std::make_shared<Tensor>(t -> getShape(), 0.0f, t -> requires_grad);

        t -> copyToDevice();
        result -> allocateDevice();

        sigmoidKernel << <getGrid(size), 256 >> > (t -> devicePtr(), result -> devicePtr(), size);
        hipDeviceSynchronize();

        result -> copyToHost();

        if (t -> requires_grad)
        {
            auto t_ptr = t -> shared_from_this();

            auto result_ptr = result;
            result -> setGradFn([t_ptr, result_ptr]() 
            {
                for (size_t i = 0; i < result_ptr -> grad.size(); ++i)
                {
                    float sig = result_ptr -> data[i];
                    t_ptr -> grad[i] += sig * (1 - sig) * result_ptr -> grad[i];
                }
            }, { t_ptr });
        }

        return result;
    }

    std::shared_ptr<Tensor> tanh_cuda(const std::shared_ptr<Tensor>& t)
    {
        int size = static_cast<int>(t -> getData().size());
        auto result = std::make_shared<Tensor>(t -> getShape(), 0.0f, t -> requires_grad);

        t -> copyToDevice();
        result -> allocateDevice();

        tanhKernel << <getGrid(size), 256 >> > (t -> devicePtr(), result -> devicePtr(), size);
        hipDeviceSynchronize();

        result->copyToHost();

        if (t->requires_grad)
        {
            auto t_ptr = t -> shared_from_this();
            auto result_ptr = result;
            result -> setGradFn([t_ptr, result_ptr]() 
            {
                for (size_t i = 0; i < result_ptr -> grad.size(); ++i)
                {
                    float tanhval = result_ptr -> data[i];
                    t_ptr -> grad[i] += (1.0f - tanhval * tanhval) * result_ptr -> grad[i];
                }
            }, { t_ptr });
        }

        return result;
    }

    // Softmax i�in hem GPU'da max bulmak hem normalize etmek i�in iki kernel, CPU yard�m� kullan�yoruz
    std::shared_ptr<Tensor> softmax_cuda(const std::shared_ptr<Tensor>& t)
    {
        int size = static_cast<int>(t -> getData().size());
        auto result = std::make_shared<Tensor>(t->getShape(), 0.0f, t -> requires_grad);

        float maxVal = *std::max_element(t -> getData().begin(), t -> getData().end());
        // CPU'da max bulmak �imdilik daha kolay

        t -> copyToDevice();
        result -> allocateDevice();

        // exp(x - max)
        softmaxExpShiftKernel << <getGrid(size), 256 >> > (t -> devicePtr(), result -> devicePtr(), maxVal, size);
        hipDeviceSynchronize();

        result -> copyToHost();

        float sumExp = 0.0f;
        for (auto val : result -> getData())
            sumExp += val;

        result -> copyToDevice();
        softmaxNormalizeKernel << <getGrid(size), 256 >> > (result->devicePtr(), sumExp, size);
        hipDeviceSynchronize();

        result -> copyToHost();

        // backward: sadece softmax'�n kendi gradyan�
        if (t -> requires_grad)
        {
            auto t_ptr = t -> shared_from_this();

            auto result_ptr = result;
            result->setGradFn([t_ptr, result_ptr]() 
            {
                for (size_t i = 0; i < result_ptr -> grad.size(); ++i)
                {
                    float s = result_ptr -> data[i];
                    t_ptr -> grad[i] += s * (1 - s) * result_ptr -> grad[i]; // simplification
                }
            }, { t_ptr });
        }

        return result;
    }

    std::shared_ptr<Tensor> matmul_cuda(const std::shared_ptr<Tensor>& a, const std::shared_ptr<Tensor>& b)
    {
        assert(a -> getShape().size() == 2 && b -> getShape().size() == 2);
        assert(a -> getShape()[1] == b -> getShape()[0]);

        int M = a -> getShape()[0];
        int K = a -> getShape()[1];
        int N = b -> getShape()[1];

        bool req_grad = a -> requires_grad || b -> requires_grad;
        auto result = std::make_shared<Tensor>(std::vector<int>{M, N}, 0.0f, req_grad);

        // GPU'ya kopya
        a -> copyToDevice();
        b -> copyToDevice();
        result -> allocateDevice();

        dim3 threads(16, 16);
        dim3 blocks((N + 15) / 16, (M + 15) / 16);

        matmulKernel << <blocks, threads >> > (a->devicePtr(), b->devicePtr(), result->devicePtr(), M, K, N);
        hipDeviceSynchronize();

        result -> copyToHost();

        if (req_grad)
        {
            auto a_ptr = a -> shared_from_this();
            auto b_ptr = b -> shared_from_this();

            auto result_ptr = result;
            result -> setGradFn([a_ptr, b_ptr, result_ptr, M, K, N]() 
            {
                // CPU-side backward: brute-force, aynen ops.cpp ile ayn�
                for (int i = 0; i < M; ++i)
                {
                    for (int l = 0; l < K; ++l)
                    {
                        float gradA = 0.0f;

                        for (int j = 0; j < N; ++j)
                            gradA += b_ptr->at({ l, j }) * result_ptr->grad[i * N + j];

                        a_ptr -> grad[i * K + l] += gradA;
                    }
                }
                for (int l = 0; l < K; ++l)
                {
                    for (int j = 0; j < N; ++j)
                    {
                        float gradB = 0.0f;

                        for (int i = 0; i < M; ++i)
                            gradB += a_ptr -> at({ i, l }) * result_ptr -> grad[i * N + j];

                        b_ptr -> grad[l * N + j] += gradB;
                    }
                }
            }, { a_ptr, b_ptr });
        }

        return result;
    }
}
